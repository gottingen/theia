#include "hip/hip_runtime.h"
// Copyright 2024 The Turbo Authors
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     https://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.
//

#include <hip/hip_complex.h>
#include <hip/hip_runtime.h>
#include <theia/theia.h>
#define USE_THEIA_CUDA_COPY_HELPERS
#include <theia/fg/compute_copy.h>
#include <cstdio>
#include <iostream>

const unsigned DIMX = 1000;
const unsigned DIMY = 800;

static const float ZMIN = 0.1f;
static const float ZMAX = 10.f;

const float DX     = 0.005f;
const size_t ZSIZE = (size_t)((ZMAX - ZMIN) / DX + 1);

void kernel(float t, float dx, float* dev_out);

int main(void) {
    float* dev_out;

    /*
     * First theia call should be a window creation call
     * so that necessary OpenGL context is created for any
     * other theia::* object to be created successfully
     */
    theia::Window wnd(DIMX, DIMY, "Three dimensional line plot demo");
    wnd.makeCurrent();

    theia::Chart chart(FG_CHART_3D);

    chart.setAxesLabelFormat("%3.1f", "%3.1f", "%.2e");

    chart.setAxesLimits(-1.1f, 1.1f, -1.1f, 1.1f, 0.f, 10.f);

    chart.setAxesTitles("x-axis", "y-axis", "z-axis");

    theia::Plot plot3 = chart.plot(ZSIZE, theia::f32);

    static float t = 0;
    THEIA_CUDA_CHECK(hipMalloc((void**)&dev_out, ZSIZE * 3 * sizeof(float)));
    kernel(t, DX, dev_out);

    GfxHandle* handle;
    createGLBuffer(&handle, plot3.vertices(), THEIA_VERTEX_BUFFER);

    /* copy your data into the vertex buffer object exposed by
     * theia::Plot class and then proceed to rendering.
     * To help the users with copying the data from compute
     * memory to display memory, theia provides copy headers
     * along with the library to help with this task
     */
    copyToGLBuffer(handle, (ComputeResourceHandle)dev_out,
                   plot3.verticesSize());

    do {
        t += 0.01f;
        kernel(t, DX, dev_out);
        copyToGLBuffer(handle, (ComputeResourceHandle)dev_out,
                       plot3.verticesSize());
        wnd.draw(chart);
    } while (!wnd.close());

    THEIA_CUDA_CHECK(hipFree(dev_out));
    releaseGLBuffer(handle);
    return 0;
}

__global__ void generateCurve(float t, float dx, float* out, const float ZMIN,
                              const size_t ZSIZE) {
    int offset = blockIdx.x * blockDim.x + threadIdx.x;

    float z = ZMIN + offset * dx;
    if (offset < ZSIZE) {
        out[3 * offset]     = cos(z * t + t) / z;
        out[3 * offset + 1] = sin(z * t + t) / z;
        out[3 * offset + 2] = z + 0.1 * sin(t);
    }
}

inline int divup(int a, int b) { return (a + b - 1) / b; }

void kernel(float t, float dx, float* dev_out) {
    static const dim3 threads(1024);
    dim3 blocks(divup(ZSIZE, 1024));

    // clang-format off
    generateCurve<<<blocks, threads>>>(t, dx, dev_out, ZMIN, ZSIZE);
    // clang-format on
}
