#include "hip/hip_runtime.h"
// Copyright 2024 The Turbo Authors
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     https://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.
//

#include <hip/hip_complex.h>
#include <hip/hip_runtime.h>
#include <theia/theia.h>
#define USE_THEIA_CUDA_COPY_HELPERS
#include <theia/fg/compute_copy.h>
#include <cstdio>
#include <iostream>

const float XMIN = -8.0f;
const float XMAX = 8.f;
const float YMIN = -8.0f;
const float YMAX = 8.f;

const float DX     = 0.5;
const size_t XSIZE = (size_t)((XMAX - XMIN) / DX);
const size_t YSIZE = (size_t)((YMAX - YMIN) / DX);

void kernel(float dx, float* dev_out);

int main(void) {
    float* dev_out;

    /*
     * First theia call should be a window creation call
     * so that necessary OpenGL context is created for any
     * other theia::* object to be created successfully
     */
    theia::Window wnd(1024, 768, "3d Surface Demo");
    wnd.makeCurrent();

    theia::Chart chart(FG_CHART_3D);
    chart.setAxesLimits(-10.f, 10.f, -10.f, 10.f, -0.5f, 1.f);
    chart.setAxesTitles("x-axis", "y-axis", "z-axis");

    theia::Surface surf = chart.surface(XSIZE, YSIZE, theia::f32);
    surf.setColor(FG_YELLOW);

    THEIA_CUDA_CHECK(
        hipMalloc((void**)&dev_out, XSIZE * YSIZE * 3 * sizeof(float)));
    kernel(DX, dev_out);

    GfxHandle* handle;
    createGLBuffer(&handle, surf.vertices(), THEIA_VERTEX_BUFFER);
    /* copy your data into the vertex buffer object exposed by
     * theia::Plot class and then proceed to rendering.
     * To help the users with copying the data from compute
     * memory to display memory, theia provides copy headers
     * along with the library to help with this task
     */
    copyToGLBuffer(handle, (ComputeResourceHandle)dev_out, surf.verticesSize());

    do { wnd.draw(chart); } while (!wnd.close());

    THEIA_CUDA_CHECK(hipFree(dev_out));
    releaseGLBuffer(handle);
    return 0;
}

__global__ void sincos_surf(float dx, float* out, const float XMIN,
                            const float YMIN, const size_t XSIZE,
                            const size_t YSIZE) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    float x = XMIN + i * dx;
    float y = YMIN + j * dx;
    if (i < XSIZE && j < YSIZE) {
        int offset          = j + i * YSIZE;
        out[3 * offset]     = x;
        out[3 * offset + 1] = y;
        float z             = sqrt(x * x + y * y) + 2.2204e-16;
        out[3 * offset + 2] = sinf(z) / z;
    }
}

inline int divup(int a, int b) { return (a + b - 1) / b; }

void kernel(float dx, float* dev_out) {
    static const dim3 threads(8, 8);
    dim3 blocks(divup(XSIZE, threads.x), divup(YSIZE, threads.y));

    // clang-format off
    sincos_surf<<<blocks, threads>>>(dx, dev_out, XMIN, YMIN, XSIZE, YSIZE);
    // clang-format on
}
