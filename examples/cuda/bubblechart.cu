#include "hip/hip_runtime.h"
// Copyright 2024 The Turbo Authors
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     https://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.
//

#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <theia/theia.h>
#define USE_THEIA_CUDA_COPY_HELPERS
#include <theia/fg/compute_copy.h>
#include <cstdio>
#include <iostream>

const unsigned DIMX = 1000;
const unsigned DIMY = 800;

static const float DX           = 0.1f;
static const float FRANGE_START = 0.f;
static const float FRANGE_END   = 2 * 3.141592f;
static const size_t DATA_SIZE   = (size_t)((FRANGE_END - FRANGE_START) / DX);

hiprandState_t* state;

void kernel(float* dev_out, int functionCode, float* colors, float* alphas,
            float* radii);

inline int divup(int a, int b) { return (a + b - 1) / b; }

__global__ void setupRandomKernel(hiprandState* states,
                                  unsigned long long seed) {
    unsigned tid = blockDim.x * blockIdx.x + threadIdx.x;
    hiprand_init(seed, tid, 0, &states[tid]);
}

int main(void) {
    THEIA_CUDA_CHECK(
        hipMalloc((void**)&state, DATA_SIZE * sizeof(hiprandState_t)));
    // clang-format off
    setupRandomKernel<<<divup(DATA_SIZE, 32), 32>>>(state, 314567);
    // clang-format on

    float* cos_out;
    float* tan_out;
    float* colors_out;
    float* alphas_out;
    float* radii_out;

    THEIA_CUDA_CHECK(
        hipMalloc((void**)&cos_out, sizeof(float) * DATA_SIZE * 2));
    THEIA_CUDA_CHECK(
        hipMalloc((void**)&tan_out, sizeof(float) * DATA_SIZE * 2));
    THEIA_CUDA_CHECK(
        hipMalloc((void**)&colors_out, sizeof(float) * DATA_SIZE * 3));
    THEIA_CUDA_CHECK(
        hipMalloc((void**)&alphas_out, sizeof(float) * DATA_SIZE));
    THEIA_CUDA_CHECK(hipMalloc((void**)&radii_out, sizeof(float) * DATA_SIZE));

    /*
     * First theia call should be a window creation call
     * so that necessary OpenGL context is created for any
     * other theia::* object to be created successfully
     */
    theia::Window wnd(DIMX, DIMY, "Bubble chart with Transparency Demo");
    wnd.makeCurrent();

    theia::Chart chart(FG_CHART_2D);
    chart.setAxesLimits(FRANGE_START, FRANGE_END, -1.0f, 1.0f);

    /* Create several plot objects which creates the necessary
     * vertex buffer objects to hold the different plot types
     */
    theia::Plot plt1 =
        chart.plot(DATA_SIZE, theia::f32, FG_PLOT_LINE, FG_MARKER_TRIANGLE);
    theia::Plot plt2 =
        chart.plot(DATA_SIZE, theia::f32, FG_PLOT_LINE, FG_MARKER_CIRCLE);

    /* Set plot colors */
    plt1.setColor(FG_RED);
    plt2.setColor(FG_GREEN);  // use a theia predefined color
    /* Set plot legends */
    plt1.setLegend("Cosine");
    plt2.setLegend("Tangent");
    /* set plot global marker size */
    plt1.setMarkerSize(20);
    /* copy your data into the opengl buffer object exposed by
     * theia::Plot class and then proceed to rendering.
     * To help the users with copying the data from compute
     * memory to display memory, theia provides copy headers
     * along with the library to help with this task
     */

    GfxHandle* handles[5];

    // create GL-CUDA interop buffers
    createGLBuffer(&handles[0], plt1.vertices(), THEIA_VERTEX_BUFFER);
    createGLBuffer(&handles[1], plt2.vertices(), THEIA_VERTEX_BUFFER);
    createGLBuffer(&handles[2], plt2.colors(), THEIA_VERTEX_BUFFER);
    createGLBuffer(&handles[3], plt2.alphas(), THEIA_VERTEX_BUFFER);
    createGLBuffer(&handles[4], plt2.radii(), THEIA_VERTEX_BUFFER);

    kernel(cos_out, 0, NULL, NULL, NULL);
    kernel(tan_out, 1, colors_out, alphas_out, radii_out);

    // copy the data from compute buffer to graphics buffer
    copyToGLBuffer(handles[0], (ComputeResourceHandle)cos_out,
                   plt1.verticesSize());
    copyToGLBuffer(handles[1], (ComputeResourceHandle)tan_out,
                   plt2.verticesSize());

    /* update color value for tan graph */
    copyToGLBuffer(handles[2], (ComputeResourceHandle)colors_out,
                   plt2.colorsSize());
    /* update alpha values for tan graph */
    copyToGLBuffer(handles[3], (ComputeResourceHandle)alphas_out,
                   plt2.alphasSize());
    /* update marker sizes for tan graph markers */
    copyToGLBuffer(handles[4], (ComputeResourceHandle)radii_out,
                   plt2.radiiSize());

    do { wnd.draw(chart); } while (!wnd.close());

    // destroy GL-CUDA Interop buffer
    releaseGLBuffer(handles[0]);
    releaseGLBuffer(handles[1]);
    releaseGLBuffer(handles[2]);
    releaseGLBuffer(handles[3]);
    releaseGLBuffer(handles[4]);
    // destroy CUDA handles
    THEIA_CUDA_CHECK(hipFree(cos_out));
    THEIA_CUDA_CHECK(hipFree(tan_out));
    THEIA_CUDA_CHECK(hipFree(colors_out));
    THEIA_CUDA_CHECK(hipFree(alphas_out));
    THEIA_CUDA_CHECK(hipFree(radii_out));

    return 0;
}

__global__ void mapKernel(float* out, int functionCode, float frange_start,
                          float dx) {
    int id  = blockIdx.x * blockDim.x + threadIdx.x;
    float x = frange_start + id * dx;
    float y;

    switch (functionCode) {
        case 0: y = cos(x); break;
        case 1: y = tan(x); break;
        default: y = sin(x); break;
    }

    out[2 * id + 0] = x;
    out[2 * id + 1] = y;
}

__global__ void colorsKernel(float* colors, hiprandState* states) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    colors[3 * id + 0] = hiprand_uniform(&states[id]);
    colors[3 * id + 1] = hiprand_uniform(&states[id]);
    colors[3 * id + 2] = hiprand_uniform(&states[id]);
}

__global__ void randKernel(float* out, hiprandState* states, float min,
                           float scale) {
    int id  = blockIdx.x * blockDim.x + threadIdx.x;
    out[id] = hiprand_uniform(&states[id]) * scale + min;
}

void kernel(float* dev_out, int functionCode, float* colors, float* alphas,
            float* radii) {
    static const dim3 threads(32);
    dim3 blocks(divup(DATA_SIZE, 32));

    // clang-format off
    mapKernel<<<blocks, threads>>>(dev_out, functionCode, FRANGE_START, DX);

    if (colors) colorsKernel<<<blocks, threads>>>(colors, state);

    if (alphas) randKernel<<<blocks, threads>>>(alphas, state, 0, 1);

    if (radii) randKernel<<<blocks, threads>>>(radii, state, 20, 60);
    // clang-format on
}
