#include "hip/hip_runtime.h"
/*******************************************************
 * Copyright (c) 2015-2019, ArrayFire
 * All rights reserved.
 *
 * This file is distributed under 3-clause BSD license.
 * The complete license agreement can be obtained at:
 * http://arrayfire.com/licenses/BSD-3-Clause
 ********************************************************/

#include <hip/hip_runtime.h>
#include <theia/theia.h>
#define USE_THEIA_CUDA_COPY_HELPERS
#include <theia/fg/compute_copy.h>

#define PI 3.14159265359

const unsigned DIMX      = 640;
const unsigned DIMY      = 480;
const float MINIMUM      = 1.0f;
const float MAXIMUM      = 20.f;
const float STEP         = 2.0f;
const float NELEMS       = (MAXIMUM - MINIMUM + 1) / STEP;
const unsigned DPOINTS[] = {5, 5, 5, 15, 15, 5, 15, 15};

void generatePoints(float* points, float* dirs);

inline int divup(int a, int b) { return (a + b - 1) / b; }

int main(void) {
    unsigned* dpoints;
    float* points;
    float* dirs;
    /*
     * First theia call should be a window creation call
     * so that necessary OpenGL context is created for any
     * other theia::* object to be created successfully
     */
    theia::Window wnd(DIMX, DIMY, "Vector Field Demo");
    wnd.makeCurrent();

    theia::Chart chart(FG_CHART_2D);
    chart.setAxesLimits(MINIMUM - 1.0f, MAXIMUM, MINIMUM - 1.0f, MAXIMUM);
    chart.setAxesTitles("x-axis", "y-axis");

    theia::Plot divPoints =
        chart.plot(4, theia::u32, FG_PLOT_SCATTER, FG_MARKER_CIRCLE);
    divPoints.setColor(0.9f, 0.9f, 0.0f, 1.f);
    divPoints.setLegend("Convergence Points");
    divPoints.setMarkerSize(24);

    size_t npoints = (size_t)(NELEMS * NELEMS);

    theia::VectorField field =
        chart.vectorField((unsigned)(npoints), theia::f32);
    field.setColor(0.f, 0.6f, 0.3f, 1.f);

    THEIA_CUDA_CHECK(hipMalloc((void**)&dpoints, 8 * sizeof(unsigned)));
    THEIA_CUDA_CHECK(hipMalloc((void**)&points, 2 * npoints * sizeof(float)));
    THEIA_CUDA_CHECK(hipMalloc((void**)&dirs, 2 * npoints * sizeof(float)));

    GfxHandle* handles[3];

    createGLBuffer(&handles[0], divPoints.vertices(), THEIA_VERTEX_BUFFER);
    createGLBuffer(&handles[1], field.vertices(), THEIA_VERTEX_BUFFER);
    createGLBuffer(&handles[2], field.directions(), THEIA_VERTEX_BUFFER);

    THEIA_CUDA_CHECK(hipMemcpy(dpoints, DPOINTS, 8 * sizeof(unsigned),
                                hipMemcpyHostToDevice));
    generatePoints(points, dirs);

    copyToGLBuffer(handles[0], (ComputeResourceHandle)dpoints,
                   divPoints.verticesSize());

    copyToGLBuffer(handles[1], (ComputeResourceHandle)points,
                   field.verticesSize());
    copyToGLBuffer(handles[2], (ComputeResourceHandle)dirs,
                   field.directionsSize());

    do { wnd.draw(chart); } while (!wnd.close());

    // destroy GL-CUDA interop buffers
    releaseGLBuffer(handles[0]);
    releaseGLBuffer(handles[1]);
    releaseGLBuffer(handles[2]);
    // destroy CUDA handles
    THEIA_CUDA_CHECK(hipFree(dpoints));
    THEIA_CUDA_CHECK(hipFree(points));
    THEIA_CUDA_CHECK(hipFree(dirs));

    return 0;
}

__global__ void pointGenKernel(float* points, float* dirs, int nelems,
                               float minimum, float step) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int j = blockDim.y * blockIdx.y + threadIdx.y;

    if (i < nelems && j < nelems) {
        int id = i + j * nelems;

        float x = minimum + i * step;
        float y = minimum + j * step;

        points[2 * id + 0] = x;
        points[2 * id + 1] = y;

        dirs[2 * id + 0] = sinf(2.0f * PI * x / 10.f);
        dirs[2 * id + 1] = sinf(2.0f * PI * y / 10.f);
    }
}

void generatePoints(float* points, float* dirs) {
    static dim3 threads(8, 8);
    dim3 blocks(divup((int)(NELEMS), threads.x),
                divup((int)(NELEMS), threads.y));

    // clang-format off
    pointGenKernel<<<blocks, threads>>>(points, dirs, (int)(NELEMS), MINIMUM,
                                        STEP);
    // clang-format on
}
